#include "hip/hip_runtime.h"
#include <assert.h>
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>

#include "extern.h"

#define TB 128
#define CUDA_ASSERT(x) (assert((x) == hipSuccess))

__global__ void rank(float *X, int *Y, int *P, int *S)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	float D[NUM_NEIGHBORS * 2];

	if (id < NUM_PAIRS) {
		int attr1 = P[id * 2];
		int attr2 = P[id * 2 + 1];
		int score = 0;
		for (int i = 0; i < NUM_EXAMPLES; i++) {
			for (int k = 0; k < NUM_NEIGHBORS; k++) {
				D[k * 2] = 2e38;
			}
			for (int j = 0; j < NUM_EXAMPLES; j++) {
				if (i != j) {
					int max_k = 0;
					for (int k = 1; k < NUM_NEIGHBORS; k++) {
						if (D[k * 2] > D[max_k * 2]) {
							max_k = k;
						}
					}
					float d1 = X[i * NUM_ATTRS + attr1] - X[j * NUM_ATTRS + attr1];
					float d2 = X[i * NUM_ATTRS + attr2] - X[j * NUM_ATTRS + attr2];
					float dist = sqrtf(d1 * d1 + d2 * d2);
					if (dist < D[max_k * 2]) {
						D[max_k * 2] = dist;
						D[max_k * 2 + 1] = Y[j];
					}
				}
			}
			for (int k = 0; k < NUM_NEIGHBORS; k++) {
				if (Y[i] == D[k * 2 + 1]) {
					score++;
				}
			}
		}
		S[id] = score;
	}
}

int main(void)
{
	int Xfd, Yfd, Pfd, *hY, *hP, *dY, *dP, *hS, *dS;
	float *hX, *dX;
	
	Xfd = open("data/X.bin", O_RDONLY);
	Yfd = open("data/Y.bin", O_RDONLY);
	Pfd = open("data/P.bin", O_RDONLY);

	hX = (float *)mmap(NULL, NUM_EXAMPLES * NUM_ATTRS * sizeof(float), PROT_READ, MAP_SHARED, Xfd, 0);
	hY = (int *)mmap(NULL, NUM_EXAMPLES * sizeof(int), PROT_READ, MAP_SHARED, Yfd, 0);
	hP = (int *)mmap(NULL, NUM_PAIRS * 2 * sizeof(int), PROT_READ, MAP_SHARED, Pfd, 0);
	hS = (int *)malloc(NUM_PAIRS * sizeof(int));

	CUDA_ASSERT(hipMalloc(&dX, NUM_EXAMPLES * NUM_ATTRS * sizeof(float)));
	CUDA_ASSERT(hipMalloc(&dY, NUM_EXAMPLES * sizeof(int)));
	CUDA_ASSERT(hipMalloc(&dP, NUM_PAIRS * 2 * sizeof(int)));
	CUDA_ASSERT(hipMalloc(&dS, NUM_PAIRS * sizeof(int)));

	CUDA_ASSERT(hipMemcpy(dX, hX, NUM_EXAMPLES * NUM_ATTRS * sizeof(float), hipMemcpyHostToDevice));
	CUDA_ASSERT(hipMemcpy(dY, hY, NUM_EXAMPLES * sizeof(int), hipMemcpyHostToDevice));
	CUDA_ASSERT(hipMemcpy(dP, hP, NUM_PAIRS * 2 * sizeof(int), hipMemcpyHostToDevice));

	rank<<<(NUM_PAIRS - 1) / TB + 1, TB>>>(dX, dY, dP, dS);
	CUDA_ASSERT(hipMemcpy(hS, dS, NUM_PAIRS * sizeof(int), hipMemcpyDeviceToHost));
	for (int i = 0; i < NUM_PAIRS; i++) {
		printf("%d\n", hS[i]);
	}
	
	return 0;
}
